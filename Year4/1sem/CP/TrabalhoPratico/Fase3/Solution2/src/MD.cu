/*
 MD.c - a simple molecular dynamics program for simulating real gas properties of Lennard-Jones particles.
 
 Copyright (C) 2016  Jonathan J. Foley IV, Chelsea Sweet, Oyewumi Akinfenwa
 
 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License as published by
 the Free Software Foundation, either version 3 of the License, or
 (at your option) any later version.
 
 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.
 
 You should have received a copy of the GNU General Public License
 along with this program.  If not, see <http://www.gnu.org/licenses/>.
 
 Electronic Contact:  foleyj10@wpunj.edu
 Mail Contact:   Prof. Jonathan Foley
 Department of Chemistry, William Paterson University
 300 Pompton Road
 Wayne NJ 07470
 
 */
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>
#include <iostream>
#include <hip/hip_runtime.h>


// Number of particles
int N = 5000;
int numThreads = 16;
int numBlocks = (N + (numThreads*2) - 1) / (numThreads * 2);

//  Lennard-Jones parameters in natural units!
double sigma = 1.;
double epsilon = 1.;
double m = 1.;
double kB = 1.;
double PE = 0.;
double mvs = 0.;
double KE = 0.;

double NA = 6.022140857e23;
double kBSI = 1.38064852e-23;  // m^2*kg/(s^2*K)

//  Size of box, which will be specified in natural units
double L;

//  Initial Temperature in Natural Units
double Tinit;  //2;
//  Vectors!
//
const int MAXPART=20001;
//  Position
double r[MAXPART][3];
//  Velocity
double v[MAXPART][3];
//  Acceleration
double a[MAXPART][3];


// atom type
char atype[10];
//  Function prototypes
//  initialize positions on simple cubic lattice, also calls function to initialize velocities
void initialize();  
//  update positions and velocities using Velocity Verlet algorithm 
//  print particle coordinates to file for rendering via VMD or other animation software
//  return 'instantaneous pressure'
double VelocityVerlet(double dt, int iter, FILE *fp);  

//joined two computations, of potential and accelarations
//  Compute Force using F = -dV/dr
//  solve F = ma for use in Velocity Verlet
//  Compute total potential energy from particle coordinates
void joinedPotentialComputeAcc();
//void computeAccelerations();
//  Numerical Recipes function for generation gaussian distribution
double gaussdist();
//  Initialize velocities according to user-supplied initial Temperature (Tinit)
void initializeVelocities();
// Joined two computations of velocity and kinetic energy
//  Compute mean squared velocity from particle velocities
//  Compute total kinetic energy from particle mass and velocities
void joinedKineticMSV();

__device__ double ourAtomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

__device__ double ourAtomicSub(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(__longlong_as_double(assumed) - val));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

int main()
{
    
    //  variable delcarations
    int i;
    double dt, Vol, Temp, Press, Pavg, Tavg, rho;
    double VolFac, TempFac, PressFac, timefac;
    double gc, Z;
    char prefix[32], tfn[32], ofn[32], afn[32];
    FILE  *tfp, *ofp, *afp;
    
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("                  WELCOME TO WILLY P CHEM MD! threads: %d\n", numThreads);
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n  ENTER A TITLE FOR YOUR CALCULATION!\n");
    scanf("%s",prefix);
    strcpy(tfn,prefix);
    strcat(tfn,"_traj.xyz");
    strcpy(ofn,prefix);
    strcat(ofn,"_output.txt");
    strcpy(afn,prefix);
    strcat(afn,"_average.txt");
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("                  TITLE ENTERED AS '%s'\n",prefix);
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    
    /*     Table of values for Argon relating natural units to SI units:
     *     These are derived from Lennard-Jones parameters from the article
     *     "Liquid argon: Monte carlo and molecular dynamics calculations"
     *     J.A. Barker , R.A. Fisher & R.O. Watts
     *     Mol. Phys., Vol. 21, 657-673 (1971)
     *
     *     mass:     6.633e-26 kg          = one natural unit of mass for argon, by definition
     *     energy:   1.96183e-21 J      = one natural unit of energy for argon, directly from L-J parameters
     *     length:   3.3605e-10  m         = one natural unit of length for argon, directly from L-J parameters
     *     volume:   3.79499-29 m^3        = one natural unit of volume for argon, by length^3
     *     time:     1.951e-12 s           = one natural unit of time for argon, by length*sqrt(mass/energy)
     ***************************************************************************************/
    
    //  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //  Edit these factors to be computed in terms of basic properties in natural units of
    //  the gas being simulated
    
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("  WHICH NOBLE GAS WOULD YOU LIKE TO SIMULATE? (DEFAULT IS ARGON)\n");
    printf("\n  FOR HELIUM,  TYPE 'He' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR NEON,    TYPE 'Ne' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR ARGON,   TYPE 'Ar' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR KRYPTON, TYPE 'Kr' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR XENON,   TYPE 'Xe' THEN PRESS 'return' TO CONTINUE\n");
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    scanf("%s",atype);
    
    if (strcmp(atype,"He")==0) {
        
        VolFac = 1.8399744000000005e-29;
        PressFac = 8152287.336171632;
        TempFac = 10.864459551225972;
        timefac = 1.7572698825166272e-12;
        
    }
    else if (strcmp(atype,"Ne")==0) {
        
        VolFac = 2.0570823999999997e-29;
        PressFac = 27223022.27659913;
        TempFac = 40.560648991243625;
        timefac = 2.1192341945685407e-12;
        
    }
    else if (strcmp(atype,"Ar")==0) {
        
        VolFac = 3.7949992920124995e-29;
        PressFac = 51695201.06691862;
        TempFac = 142.0950000000000;
        timefac = 2.09618e-12;
        //strcpy(atype,"Ar");
        
    }
    else if (strcmp(atype,"Kr")==0) {
        
        VolFac = 4.5882712000000004e-29;
        PressFac = 59935428.40275003;
        TempFac = 199.1817584391428;
        timefac = 8.051563913585078e-13;
        
    }
    else if (strcmp(atype,"Xe")==0) {
        
        VolFac = 5.4872e-29;
        PressFac = 70527773.72794868;
        TempFac = 280.30305642163006;
        timefac = 9.018957925790732e-13;
        
    }
    else {
        
        VolFac = 3.7949992920124995e-29;
        PressFac = 51695201.06691862;
        TempFac = 142.0950000000000;
        timefac = 2.09618e-12;
        strcpy(atype,"Ar");
        
    }
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n                     YOU ARE SIMULATING %s GAS! \n",atype);
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n  YOU WILL NOW ENTER A FEW SIMULATION PARAMETERS\n");
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n\n  ENTER THE INTIAL TEMPERATURE OF YOUR GAS IN KELVIN\n");
    scanf("%lf",&Tinit);
    // Make sure temperature is a positive number!
    if (Tinit<0.) {
        printf("\n  !!!!! ABSOLUTE TEMPERATURE MUST BE A POSITIVE NUMBER!  PLEASE TRY AGAIN WITH A POSITIVE TEMPERATURE!!!\n");
        exit(0);
    }
    // Convert initial temperature from kelvin to natural units
    Tinit /= TempFac;
    
    
    printf("\n\n  ENTER THE NUMBER DENSITY IN moles/m^3\n");
    printf("  FOR REFERENCE, NUMBER DENSITY OF AN IDEAL GAS AT STP IS ABOUT 40 moles/m^3\n");
    printf("  NUMBER DENSITY OF LIQUID ARGON AT 1 ATM AND 87 K IS ABOUT 35000 moles/m^3\n");
    
    scanf("%lf",&rho);
    
    
    Vol = N/(rho*NA);
    
    Vol /= VolFac;
    
    //  Limiting N to MAXPART for practical reasons
    if (N>=MAXPART) {
        
        printf("\n\n\n  MAXIMUM NUMBER OF PARTICLES IS %i\n\n  PLEASE ADJUST YOUR INPUT FILE ACCORDINGLY \n\n", MAXPART);
        exit(0);
        
    }
    //  Check to see if the volume makes sense - is it too small?
    //  Remember VDW radius of the particles is 1 natural unit of length
    //  and volume = L*L*L, so if V = N*L*L*L = N, then all the particles
    //  will be initialized with an interparticle separation equal to 2xVDW radius
    if (Vol<N) {
        
        printf("\n\n\n  YOUR DENSITY IS VERY HIGH!\n\n");
        printf("  THE NUMBER OF PARTICLES IS %i AND THE AVAILABLE VOLUME IS %f NATURAL UNITS\n",N,Vol);
        printf("  SIMULATIONS WITH DENSITY GREATER THAN 1 PARTCICLE/(1 Natural Unit of Volume) MAY DIVERGE\n");
        printf("  PLEASE ADJUST YOUR INPUT FILE ACCORDINGLY AND RETRY\n\n");
        exit(0);
    }
    // Vol = L*L*L;
    // Length of the box in natural units:
    L = pow(Vol,(1./3));
    
    //  Files that we can write different quantities to
    tfp = fopen(tfn,"w");     //  The MD trajectory, coordinates of every particle at each timestep
    ofp = fopen(ofn,"w");     //  Output of other quantities (T, P, gc, etc) at every timestep
    afp = fopen(afn,"w");    //  Average T, P, gc, etc from the simulation
    
    int NumTime;
    if (strcmp(atype,"He")==0) {
        
        // dt in natural units of time s.t. in SI it is 5 f.s. for all other gasses
        dt = 0.2e-14/timefac;
        //  We will run the simulation for NumTime timesteps.
        //  The total time will be NumTime*dt in natural units
        //  And NumTime*dt multiplied by the appropriate conversion factor for time in seconds
        NumTime=50000;
    }
    else {
        dt = 0.5e-14/timefac;
        NumTime=200;
        
    }
    
    //  Put all the atoms in simple crystal lattice and give them random velocities
    //  that corresponds to the initial temperature we have specified
    initialize();
    
    //  Based on their positions, calculate the ininial intermolecular forces
    //  The accellerations of each particle will be defined from the forces and their
    //  mass, and this will allow us to update their positions via Newton's law
    joinedPotentialComputeAcc();
    
    // Print number of particles to the trajectory file
    fprintf(tfp,"%i\n",N);
    
    //  We want to calculate the average Temperature and Pressure for the simulation
    //  The variables need to be set to zero initially
    Pavg = 0;
    Tavg = 0;
    
    
    int tenp = floor(NumTime/10);
    fprintf(ofp,"  time (s)              T(t) (K)              P(t) (Pa)           Kinetic En. (n.u.)     Potential En. (n.u.) Total En. (n.u.)\n");
    printf("  PERCENTAGE OF CALCULATION COMPLETE:\n  [");
    for (i=0; i<NumTime+1; i++) {
        
        //  This just prints updates on progress of the calculation for the users convenience
        if (i==tenp) printf(" 10 |");
        else if (i==2*tenp) printf(" 20 |");
        else if (i==3*tenp) printf(" 30 |");
        else if (i==4*tenp) printf(" 40 |");
        else if (i==5*tenp) printf(" 50 |");
        else if (i==6*tenp) printf(" 60 |");
        else if (i==7*tenp) printf(" 70 |");
        else if (i==8*tenp) printf(" 80 |");
        else if (i==9*tenp) printf(" 90 |");
        else if (i==10*tenp) printf(" 100 ]\n");
        fflush(stdout);
        
        
        // This updates the positions and velocities using Newton's Laws
        // Also computes the Pressure as the sum of momentum changes from wall collisions / timestep
        // which is a Kinetic Theory of gasses concept of Pressure
        Press = VelocityVerlet(dt, i+1, tfp);
        Press *= PressFac;
        
        //  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
        //  Now we would like to calculate somethings about the system:
        //  Instantaneous mean velocity squared, Temperature, Pressure
        //  Potential, and Kinetic Energy
        //  We would also like to use the IGL to try to see if we can extract the gas constant
        joinedKineticMSV();
        
        // Temperature from Kinetic Theory
        Temp = m*mvs/(3*kB) * TempFac;
        
        // Instantaneous gas constant and compressibility - not well defined because
        // pressure may be zero in some instances because there will be zero wall collisions,
        // pressure may be very high in some instances because there will be a number of collisions
        gc = NA*Press*(Vol*VolFac)/(N*Temp);
        Z  = Press*(Vol*VolFac)/(N*kBSI*Temp);
        
        Tavg += Temp;
        Pavg += Press;
        
        fprintf(ofp,"  %8.4e  %20.8f  %20.8f %20.8f  %20.8f  %20.8f \n",i*dt*timefac,Temp,Press,KE, PE, KE+PE);
        
        
    }
    
    // Because we have calculated the instantaneous temperature and pressure,
    // we can take the average over the whole simulation here
    Pavg /= NumTime;
    Tavg /= NumTime;
    Z = Pavg*(Vol*VolFac)/(N*kBSI*Tavg);
    gc = NA*Pavg*(Vol*VolFac)/(N*Tavg);
    fprintf(afp,"  Total Time (s)      T (K)               P (Pa)      PV/nT (J/(mol K))         Z           V (m^3)              N\n");
    fprintf(afp," --------------   -----------        ---------------   --------------   ---------------   ------------   -----------\n");
    fprintf(afp,"  %8.4e  %15.12f       %15.12f     %10.12f       %10.12f        %10.12e         %i\n",i*dt*timefac,Tavg,Pavg,gc,Z,Vol*VolFac,N);
    
    printf("\n  TO ANIMATE YOUR SIMULATION, OPEN THE FILE \n  '%s' WITH VMD AFTER THE SIMULATION COMPLETES\n",tfn);
    printf("\n  TO ANALYZE INSTANTANEOUS DATA ABOUT YOUR MOLECULE, OPEN THE FILE \n  '%s' WITH YOUR FAVORITE TEXT EDITOR OR IMPORT THE DATA INTO EXCEL\n",ofn);
    printf("\n  THE FOLLOWING THERMODYNAMIC AVERAGES WILL BE COMPUTED AND WRITTEN TO THE FILE  \n  '%s':\n",afn);
    printf("\n  AVERAGE TEMPERATURE (K):                 %15.12f\n",Tavg);
    printf("\n  AVERAGE PRESSURE  (Pa):                  %15.12f\n",Pavg);
    printf("\n  PV/nT (J * mol^-1 K^-1):                 %15.12f\n",gc);
    printf("\n  PERCENT ERROR of pV/nT AND GAS CONSTANT: %15.12f\n",100*fabs(gc-8.3144598)/8.3144598);
    printf("\n  THE COMPRESSIBILITY (unitless):          %15.12f \n",Z);
    printf("\n  TOTAL VOLUME (m^3):                      %10.12e \n",Vol*VolFac);
    printf("\n  NUMBER OF PARTICLES (unitless):          %i \n", N);
    
    fclose(tfp);
    fclose(ofp);
    fclose(afp);
    
    return 0;
}


void initialize() {
    int n, p, i, j, k;
    double pos;
    
    // Number of atoms in each direction
    n = int(ceil(pow(N, 1.0/3)));
    
    //  spacing between atoms along a given direction
    pos = L / n;
    
    //  index for number of particles assigned positions
    p = 0;
    //  initialize positions
    for (i=0; i<n; i++) {
        for (j=0; j<n; j++) {
            for (k=0; k<n; k++) {
                if (p<N) {
                    
                    r[p][0] = (i + 0.5)*pos;
                    r[p][1] = (j + 0.5)*pos;
                    r[p][2] = (k + 0.5)*pos;
                }
                p++;
            }
        }
    }
    
    // Call function to initialize velocities
    initializeVelocities();
    
    /***********************************************
     *   Uncomment if you want to see what the initial positions and velocities are
     printf("  Printing initial positions!\n");
     for (i=0; i<N; i++) {
     printf("  %6.3e  %6.3e  %6.3e\n",r[i][0],r[i][1],r[i][2]);
     }
     
     printf("  Printing initial velocities!\n");
     for (i=0; i<N; i++) {
     printf("  %6.3e  %6.3e  %6.3e\n",v[i][0],v[i][1],v[i][2]);
     }
     */
    
    
    
}   



void joinedKineticMSV(){
    double v3_mvs = 0, v3=0; 
    KE =0.;
    for (int i=0; i<N; i++) {
        v3 =  v[i][0]*v[i][0] + v[i][1]*v[i][1] +v [i][2]*v[i][2];
        
        v3_mvs += v3; 
        KE += m*v3/2.;
    }
    mvs = v3_mvs/N;
}


__global__ void rijCalc(double *d_r, double *d_a, double *d_P,int N, double sigma, double epsilon){
    double sigma6 = sigma*sigma*sigma*sigma*sigma*sigma;
    int i = threadIdx.x + blockIdx.x * blockDim.x; // i usada para o primeiro loop
    int inv_i = N-i-1; // i usada para o segundo loop 

    double f, rSqd, term1, term2, f0, f1, f2, a0 = 0, a1 = 0, a2 = 0, P = 0, ainv_i0 = 0, ainv_i1 = 0, ainv_i2 = 0;
    double rij[3];
    if((i < (N/2))){
        for (int j = i+1; j < N; j++) {
            if(inv_i < j && inv_i >= (N/2)){ // segundo loop vai de N-i até N
                rij[0] = d_r[inv_i*3] - d_r[j*3]; 
                rij[1] = d_r[inv_i*3+1] - d_r[j*3+1];
                rij[2] = d_r[inv_i*3+2] - d_r[j*3+2];

                rSqd = sigma6 / (rij[0] * rij[0] + rij[1] * rij[1] + rij[2] * rij[2]);

                term2 = rSqd * rSqd * rSqd ;
                term1 = term2 * term2;

                P += 8*epsilon* (term1-term2);

                f = term2 * rSqd * (48 * term2 - 24);

                f0 = rij[0] * f;
                f1 = rij[1] * f;
                f2 = rij[2] * f;

                ourAtomicSub(&d_a[j*3], f0);
                ourAtomicSub(&d_a[j*3+1], f1);
                ourAtomicSub(&d_a[j*3+2], f2);
                ainv_i0 += f0;
                ainv_i1 += f1;
                ainv_i2 += f2; 
            } 

            rij[0] = d_r[i*3] - d_r[j*3]; 
            rij[1] = d_r[i*3+1] - d_r[j*3+1];
            rij[2] = d_r[i*3+2] - d_r[j*3+2];

            rSqd = sigma6 / (rij[0] * rij[0] + rij[1] * rij[1] + rij[2] * rij[2]);

            term2 = rSqd * rSqd * rSqd ;
            term1 = term2 * term2;

            P += 8*epsilon* (term1-term2);

            f = term2 * rSqd * (48 * term2 - 24);

            f0 = rij[0] * f;
            f1 = rij[1] * f;
            f2 = rij[2] * f;

            ourAtomicSub(&d_a[j*3], f0);
            ourAtomicSub(&d_a[j*3+1], f1);
            ourAtomicSub(&d_a[j*3+2], f2);
            a0 += f0;
            a1 += f1;
            a2 += f2;        
        }
        
        ourAtomicAdd(&d_a[i*3], a0);
        ourAtomicAdd(&d_a[i*3+1], a1);
        ourAtomicAdd(&d_a[i*3+2], a2);
        ourAtomicAdd(&d_a[inv_i*3], ainv_i0);
        ourAtomicAdd(&d_a[inv_i*3+1], ainv_i1);
        ourAtomicAdd(&d_a[inv_i*3+2], ainv_i2);
        ourAtomicAdd(&d_P[blockIdx.x], P);
    }
}

// Function to calculate the potential energy of the system
//  Function to compute accelarations - Uses the derivative of the Lennard-Jones potential to calculate
//  the forces on each atom.  Then uses a = F/m to calculate the accelleration of each atom. 

void joinedPotentialComputeAcc(){

    double *d_r, *d_a, *d_P;
    
    hipMalloc((void **) &d_r, 3*N*sizeof(double));
    hipMalloc((void **) &d_a, 3*N*sizeof(double));
    hipMalloc((void **) &d_P, numBlocks*sizeof(double));
    
    hipMemcpy(d_r, &r, 3*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_a, 0, 3*N*sizeof(double));
    hipMemset(d_P, 0, numBlocks*sizeof(double));

    dim3 grid_size(numBlocks); 
    dim3 block_size(numThreads);

    rijCalc<<<grid_size, block_size>>>(d_r, d_a, d_P, N, sigma, epsilon);

    double P[numBlocks];
    hipMemcpy(P, d_P, numBlocks*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(a, d_a, 3*N*sizeof(double), hipMemcpyDeviceToHost);

    PE = 0.;
    for(int i = 0; i < numBlocks; i++){
        PE += P[i];
    }

    hipFree(d_r);
    hipFree(d_P);
    hipFree(d_a);
}

// returns sum of dv/dt*m/A (aka Pressure) from elastic collisions with walls
double VelocityVerlet(double dt, int iter, FILE *fp) {
    int i;
    
    //  Compute accelerations from forces at current position
    // this call was removed (commented) for predagogical reasons
    //  Update positions and velocity with current velocity and acceleration
    //printf("  Updated Positions!\n");
    for (i=0; i<N; i++) {
            r[i][0] += (v[i][0] + 0.5*a[i][0]*dt)*dt;
            r[i][1] += (v[i][1] + 0.5*a[i][1]*dt)*dt;
            r[i][2] += (v[i][2] + 0.5*a[i][2]*dt)*dt;

            v[i][0] += 0.5*a[i][0]*dt;
            v[i][1] += 0.5*a[i][1]*dt;
            v[i][2] += 0.5*a[i][2]*dt;
        //printf("  %i  %6.4e   %6.4e   %6.4e\n",i,r[i][0],r[i][1],r[i][2]);
    }
    //  Update accellerations from updated positions
    joinedPotentialComputeAcc();
    //  Update velocity with updated acceleration
    for (i=0; i<N; i++) {
        
        v[i][0] += 0.5*a[i][0]*dt;
        v[i][1] += 0.5*a[i][1]*dt;
        v[i][2] += 0.5*a[i][2]*dt;
    }
    

    double psum = 0.;
    // Elastic walls
    for (i=0; i<N; i++) {
        if (r[i][0]<0. || r[i][0]>=L) {
                v[i][0] *=-1.; //- elastic walls
                psum += 2*m*fabs(v[i][0])/dt; // contribution to pressure from "left" walls
        }

        if (r[i][1]<0. || r[i][1]>=L) {
            v[i][1] *=-1.; //- elastic walls
            psum += 2*m*fabs(v[i][1])/dt; // contribution to pressure from "left" walls
        }

        if (r[i][2]<0. || r[i][2]>=L) {
            v[i][2] *=-1.; //- elastic walls
            psum += 2*m*fabs(v[i][2])/dt; // contribution to pressure from "left" walls
        }
    }
    
    
    /* removed, uncomment to save atoms positions */
    /*for (i=0; i<N; i++) {
        fprintf(fp,"%s",atype);
        for (j=0; j<3; j++) {
            fprintf(fp,"  %12.10e ",r[i][j]);
        }
        fprintf(fp,"\n");
    }*/
    //fprintf(fp,"\n \n");
    
    return psum/(6*L*L);
}


void initializeVelocities() {
    
    int i;
    // Vcm = sum_i^N  m*v_i/  sum_i^N  M
    // Compute center-of-mas velocity according to the formula above
    double vCM[3] = {0, 0, 0};

    for (i=0; i<N; i++) {
        v[i][0] = gaussdist();
        v[i][1] = gaussdist();
        v[i][2] = gaussdist();

        vCM[0] += m*v[i][0];
        vCM[1] += m*v[i][1];
        vCM[2] += m*v[i][2];
    }


    vCM[0] /= N*m;
    vCM[1] /= N*m;
    vCM[2] /= N*m;

    
    double vSqdSum, lambda;
    vSqdSum=0.;
    for (i=0; i<N; i++) {
        //  Subtract out the center-of-mass velocity from the
        //  velocity of each particle... effectively set the
        //  center of mass velocity to zero so that the system does
        //  not drift in space!

        v[i][0] -= vCM[0];
        v[i][1] -= vCM[1];
        v[i][2] -= vCM[2];

        //  Now we want to scale the average velocity of the system
        //  by a factor which is consistent with our initial temperature, Tinit
        vSqdSum += v[i][0]*v[i][0];
        vSqdSum += v[i][1]*v[i][1];
        vSqdSum += v[i][2]*v[i][2];
    }

    lambda = sqrt( 3*(N-1)*Tinit/vSqdSum);
    
    for (i=0; i<N; i++) {
        v[i][0] *= lambda;
        v[i][1] *= lambda;
        v[i][2] *= lambda;
    }
}

//  Numerical recipes Gaussian distribution number generator
double gaussdist() {
    static bool available = false;
    static double gset;
    double fac, rsq, v1, v2;
    if (!available) {
        do {
            v1 = 2.0 * rand() / double(RAND_MAX) - 1.0;
            v2 = 2.0 * rand() / double(RAND_MAX) - 1.0;
            rsq = v1 * v1 + v2 * v2;
        } while (rsq >= 1.0 || rsq == 0.0);
        
        fac = sqrt(-2.0 * log(rsq) / rsq);
        gset = v1 * fac;
        available = true;
        
        return v2*fac;
    } else {
        
        available = false;
        return gset;
        
    }
}
